#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Solver.h"

#define BLOCKS 1
#define THREADS 20

// poisson solver

__global__ void solve_poisson(float ps[Nxb+2][Nyb+2],float ps_old[Nxb+2][Nyb+2],float gr_dx_centers[Nxb+2][Nyb+2],float gr_dy_centers[Nxb+2][Nyb+2],
                         float gr_dx_nodes[Nxb+2][Nyb+2],float gr_dy_nodes[Nxb+2][Nyb+2],float ps_RHS[Nxb][Nyb])
{
   int i,j;

   j = (blockIdx.x * blockDim.x) + threadIdx.x;
   i = (blockIdx.y * blockDim.y) + threadIdx.y;

  if(i>=1 && i<=Nxb && j>=1 && j<=Nyb) 

  {

   ps[i][j]=((ps_old[i][j+1]/(gr_dy_centers[i][j]*gr_dy_nodes[i][j]))+(ps[i][j-1]/(gr_dy_nodes[i][j]*gr_dy_centers[i-1][j-1]))
            +(ps_old[i+1][j]/(gr_dx_centers[i][j]*gr_dx_nodes[i][j]))+(ps[i-1][j]/(gr_dx_nodes[i][j]*gr_dx_centers[i-1][j-1]))
           +ps_RHS[i-1][j-1])
           *(1/((1/(gr_dx_nodes[i][j]*gr_dx_centers[i-1][j-1]))+(1/(gr_dy_nodes[i][j]*gr_dy_centers[i-1][j-1]))+
                (1/(gr_dx_nodes[i][j]*gr_dx_centers[i][j]))+(1/(gr_dy_nodes[i][j]*gr_dy_centers[i][j]))));

   }
   
}

// function called from main fortran program
extern "C" void poisson_kernel_wrapper_(float *ps,float *ps_old,float *gr_dx_centers,float *gr_dy_centers,float *gr_dx_nodes,float *gr_dy_nodes,float *ps_RHS)
{
   float  *ps_d,*ps_old_d,*gr_dx_centers_d,*gr_dy_centers_d,*gr_dx_nodes_d,*gr_dy_nodes_d,*ps_RHS_d; // declare GPU vector copies

   dim3 numBlocks(BLOCKS,BLOCKS);
   dim3 threadsPerBlock(THREADS,THREADS);

   // Allocate memory on GPU
   hipMalloc( (void **)&ps_d, sizeof(float) * (Nxb+2) * (Nyb+2) );
   hipMalloc( (void **)&ps_old_d, sizeof(float) * (Nxb+2) * (Nyb+2) );
   hipMalloc( (void **)&gr_dx_centers_d, sizeof(float) * (Nxb+2) * (Nyb+2) );
   hipMalloc( (void **)&gr_dy_centers_d, sizeof(float) * (Nxb+2) * (Nyb+2) );
   hipMalloc( (void **)&gr_dx_nodes_d, sizeof(float) * (Nxb+2) * (Nyb+2) );
   hipMalloc( (void **)&gr_dy_nodes_d, sizeof(float) * (Nxb+2) * (Nyb+2) );
   hipMalloc( (void **)&ps_RHS_d, sizeof(float) * (Nxb) * (Nyb) );

   // copy vectors from CPU to GPU

   hipMemcpy( ps_d, ps, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyHostToDevice );
   hipMemcpy( ps_old_d, ps_old, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyHostToDevice );
   hipMemcpy( gr_dx_centers_d, gr_dx_centers, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyHostToDevice );
   hipMemcpy( gr_dy_centers_d, gr_dy_centers, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyHostToDevice );
   hipMemcpy( gr_dx_nodes_d, gr_dx_nodes, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyHostToDevice );
   hipMemcpy( gr_dy_nodes_d, gr_dy_nodes, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyHostToDevice );
   hipMemcpy( ps_RHS_d, ps_RHS, sizeof(float) * (Nxb)*(Nyb), hipMemcpyHostToDevice );

   // call function on GPU
   solve_poisson<<<numBlocks, threadsPerBlock>>>((float(*) [Nyb+2])ps_d,(float(*) [Nyb+2])ps_old_d,(float(*) [Nyb+2])gr_dx_centers_d,
                                            (float(*) [Nyb+2])gr_dy_centers_d,(float(*) [Nyb+2])gr_dx_nodes_d,(float(*) [Nyb+2])gr_dy_nodes_d,(float(*) [Nyb])ps_RHS_d);

   // copy vectors back from GPU to CPU

   hipMemcpy(ps, ps_d, sizeof(float) * (Nxb+2)*(Nyb+2), hipMemcpyDeviceToHost );

   // free GPU memory

   hipFree(ps_d);
   hipFree(ps_old_d);
   hipFree(gr_dx_centers_d);
   hipFree(gr_dx_nodes_d);
   hipFree(gr_dy_centers_d);
   hipFree(gr_dy_nodes_d);
   hipFree(ps_RHS);

   return;
}

